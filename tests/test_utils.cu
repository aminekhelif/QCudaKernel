#include <gtest/gtest.h>
extern "C" void launchQuantumSearch(const float*, int, float, int*, hipStream_t);
extern "C" void autoTuneKernelParams();
extern "C" int getSelectedTileSize();

TEST(UtilsTest, AutoTuner) {
    autoTuneKernelParams();
    int tile = getSelectedTileSize();
    ASSERT_GT(tile,0);
}

TEST(UtilsTest, QuantumSearch) {
    int size=1024;
    std::vector<float> data(size);
    for(int i=0;i<size;i++) data[i]=(float)rand()/RAND_MAX;
    int specialIdx = 512;
    data[specialIdx]=0.50001f;

    float *d_data;
    int *d_result;
    hipMalloc(&d_data,size*sizeof(float));
    hipMalloc(&d_result,sizeof(int));

    hipMemcpy(d_data,data.data(),size*sizeof(float),hipMemcpyHostToDevice);
    int initVal=-1;
    hipMemcpy(d_result,&initVal,sizeof(int),hipMemcpyHostToDevice);

    launchQuantumSearch(d_data,size,0.5f,d_result,0);
    hipDeviceSynchronize();

    int found;
    hipMemcpy(&found,d_result,sizeof(int),hipMemcpyDeviceToHost);
    hipFree(d_data);
    hipFree(d_result);

    ASSERT_EQ(found,specialIdx);
}