#include <hip/hip_runtime.h>

extern "C" void runWithCudaGraph(void(*kernelFunc)(hipStream_t), hipStream_t stream=0) {
    hipGraph_t graph;
    hipGraphExec_t graphExec;

    hipStream_t capStream;
    hipStreamCreate(&capStream);

    hipStreamBeginCapture(capStream, hipStreamCaptureModeGlobal);
    kernelFunc(capStream);
    hipStreamEndCapture(capStream, &graph);
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);

    hipGraphLaunch(graphExec, stream);
    hipStreamSynchronize(stream);

    hipGraphDestroy(graph);
    hipGraphExecDestroy(graphExec);
    hipStreamDestroy(capStream);
}